
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

int const num = 512;

int * vector_add(int *a, int *b, int *c){
    for(int idx =0; idx < num; idx++){
        c[idx] = a[idx] + b[idx];
    }

    return c;
}

void vector_print(int *a, int *b, int *c){
    for(int idx =0; idx < num; idx++){
        printf("\n %d + %d = %d", a[idx], b[idx], c[idx] );
    }

}

void vector_fill( int *data){
    for(int idx =0; idx < num; idx++){
        data[idx] = idx;
    }
}


int main(){
    int *a, *b, *c;
    int size = num * sizeof(int);

    a = (int *)malloc(size);
    vector_fill(a);

    b = (int *)malloc(size);
    vector_fill(b);

    c = (int *)malloc(size);

    c = vector_add(a, b, c);

    vector_print(a, b, c);

    free(a); free(b); free(c);

    return 0;
}

